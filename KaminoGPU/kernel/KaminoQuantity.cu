# include "../include/KaminoQuantity.h"

hipChannelFormatDesc KaminoQuantity::channelFormat
= hipCreateChannelDesc(sizeof(fReal) * byte2Bits, 0, 0, 0, hipChannelFormatKindFloat);

void KaminoQuantity::bindTexture(table2D& tex)
{
	checkCudaErrors(hipBindTexture2D(0, &tex, gpuThisStep, &desc,
		nPhi, nTheta, thisStepPitch));
}

void KaminoQuantity::unbindTexture(table2D& tex)
{
	checkCudaErrors(hipUnbindTexture(tex));
}

void KaminoQuantity::copyToGPU()
{
	/* 
	Pitch : nPhi * sizeof(fReal)
	Width : nPhi * sizeof(fReal)
	Height: nTheta
	*/
	checkCudaErrors(hipMemcpy2D(gpuThisStep, thisStepPitch, cpuBuffer, 
		nPhi * sizeof(fReal), nPhi * sizeof(fReal), nTheta, hipMemcpyHostToDevice));
}

void KaminoQuantity::copyBackToCPU()
{
	checkCudaErrors(hipMemcpy2D((void*)this->cpuBuffer, nPhi * sizeof(fReal), (void*)this->gpuThisStep,
	this->thisStepPitch, nPhi * sizeof(fReal), nTheta, hipMemcpyDeviceToHost));
}

KaminoQuantity::KaminoQuantity(std::string attributeName, size_t nPhi, size_t nTheta,
	fReal phiOffset, fReal thetaOffset)
	: nPhi(nPhi), nTheta(nTheta), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen),
	attrName(attributeName), phiOffset(phiOffset), thetaOffset(thetaOffset)
{
	desc = hipCreateChannelDesc<fReal>();

	cpuBuffer = new fReal[nPhi * nTheta];
	checkCudaErrors(hipMallocPitch((void**)&gpuThisStep, &thisStepPitch, nPhi * sizeof(fReal), nTheta));
	checkCudaErrors(hipMallocPitch((void**)&gpuNextStep, &nextStepPitch, nPhi * sizeof(fReal), nTheta));
}

KaminoQuantity::~KaminoQuantity()
{
	delete[] cpuBuffer;

	checkCudaErrors(hipFree(gpuThisStep));
	checkCudaErrors(hipFree(gpuNextStep));
}

std::string KaminoQuantity::getName()
{
	return this->attrName;
}

size_t KaminoQuantity::getNPhi()
{
	return this->nPhi;
}

size_t KaminoQuantity::getNTheta()
{
	return this->nTheta;
}

void KaminoQuantity::swapGPUBuffer()
{
	fReal* tempPtr = this->gpuThisStep;
	this->gpuThisStep = this->gpuNextStep;
	this->gpuNextStep = tempPtr;
}

fReal KaminoQuantity::getCPUValueAt(size_t phi, size_t theta)
{
	return this->accessCPUValueAt(phi, theta);
}

void KaminoQuantity::setCPUValueAt(size_t phi, size_t theta, fReal val)
{
	this->accessCPUValueAt(phi, theta) = val;
}

fReal& KaminoQuantity::accessCPUValueAt(size_t phi, size_t theta)
{
	return this->cpuBuffer[getIndex(phi, theta)];
}

fReal KaminoQuantity::getThetaOffset()
{
	return this->thetaOffset;
}

fReal KaminoQuantity::getPhiOffset()
{
	return this->phiOffset;
}

fReal* KaminoQuantity::getGPUThisStep()
{
	return this->gpuThisStep;
}

fReal* KaminoQuantity::getGPUNextStep()
{
	return this->gpuNextStep;
}

size_t KaminoQuantity::getThisStepPitch()
{
	return this->thisStepPitch;
}

size_t KaminoQuantity::getNextStepPitch()
{
	return this->nextStepPitch;
}