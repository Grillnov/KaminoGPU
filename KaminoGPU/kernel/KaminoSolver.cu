#include "hip/hip_runtime.h"
# include "../include/KaminoSolver.h"
# include "../include/CubicSolver.h"

// CONSTRUCTOR / DESTRUCTOR >>>>>>>>>>

const int fftRank = 1;

KaminoSolver::KaminoSolver(size_t nPhi, size_t nTheta, fReal radius, fReal frameDuration,
	fReal A, int B, int C, int D, int E) :
	nPhi(nPhi), nTheta(nTheta), radius(radius), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen), frameDuration(frameDuration),
	timeStep(0.0), timeElapsed(0.0),
	A(A), B(B), C(C), D(D), E(E)
{
	/// Replace it later with functions from hip/hip_runtime_api.h!
	checkCudaErrors(hipSetDevice(0));

	checkCudaErrors(hipMalloc((void **)&gpuUFourier,
		sizeof(ComplexFourier) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuUReal,
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuUImag,
		sizeof(fReal) * nPhi * nTheta));

	checkCudaErrors(hipMalloc((void **)&gpuFFourier,
		sizeof(ComplexFourier) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuFReal,
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuFImag,
		sizeof(fReal) * nPhi * nTheta));

	checkCudaErrors(hipMalloc((void **)(&gpuA),
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)(&gpuB),
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)(&gpuC),
		sizeof(fReal) * nPhi * nTheta));
	precomputeABCCoef();

	this->velPhi = new KaminoQuantity("velPhi", nPhi, nTheta,
		vPhiPhiOffset, vPhiThetaOffset);
	this->velTheta = new KaminoQuantity("velTheta", nPhi, nTheta - 1,
		vThetaPhiOffset, vThetaThetaOffset);
	this->pressure = new KaminoQuantity("p", nPhi, nTheta,
		centeredPhiOffset, centeredThetaOffset);

	this->cpuGridTypesBuffer = new gridType[nPhi * nTheta];
	checkCudaErrors(hipMalloc((void **)(this->gpuGridTypes),
		sizeof(gridType) * nPhi * nTheta));

	initialize_velocity();
	copyVelocity2GPU();

	initialize_boundary();
	copyGridType2GPU();

	setTextureParams(texVelPhi);
	setTextureParams(texVelTheta);
	setTextureParams(texBeingAdvected);
	setTextureParams(texPressure);

	int sigLenArr[1];
	sigLenArr[0] = nPhi;
	checkCudaErrors(hipfftPlanMany(&kaminoPlan, fftRank, sigLenArr,
		NULL, 1, nPhi,
		NULL, 1, nPhi,
		HIPFFT_C2C, nTheta));
}

KaminoSolver::~KaminoSolver()
{
	checkCudaErrors(hipFree(gpuUFourier));
	checkCudaErrors(hipFree(gpuUReal));
	checkCudaErrors(hipFree(gpuUImag));

	checkCudaErrors(hipFree(gpuFFourier));
	checkCudaErrors(hipFree(gpuFReal));
	checkCudaErrors(hipFree(gpuFImag));
	
	checkCudaErrors(hipFree(gpuA));
	checkCudaErrors(hipFree(gpuB));
	checkCudaErrors(hipFree(gpuC));

	delete this->velPhi;
	delete this->velTheta;
	delete this->pressure;

	delete[] cpuGridTypesBuffer;
	checkCudaErrors(hipFree(gpuGridTypes));
}

void KaminoSolver::setTextureParams(table2D tex)
{
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeMirror;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = true;
}

void KaminoSolver::copyVelocity2GPU()
{
	velPhi->copyToGPU();
	velTheta->copyToGPU();
}

void KaminoSolver::bindPressure2Tex(table2D tex)
{
	this->pressure->bindTexture(tex);
}
void KaminoSolver::bindVelocity2Tex(table2D phi, table2D theta)
{
	this->velPhi->bindTexture(phi);
	this->velTheta->bindTexture(theta);
}

__global__ void precomputeABCKernel
(fReal* A, fReal* B, fReal* C, fReal gridLen, int nPhi, int nTheta)
{
	int nIndex = blockIdx.x;
	int n = nIndex - nPhi / 2;
	int i = threadIdx.x;
	int index = nIndex * nTheta + i;
	fReal thetaI = (i + centeredThetaOffset) * gridLen;

	fReal cosThetaI = cosf(thetaI);
	fReal sinThetaI = sinf(thetaI);

	if (n != 0)
	{
		A[index] = 1.0 / (gridLen * gridLen)
			- 0.5 * cosThetaI / gridLen / sinThetaI;
		B[index] = -2.0 / (gridLen * gridLen) - n * n / (sinThetaI * sinThetaI);
		C[index] = 1.0 / (gridLen * gridLen) + 0.5 * cosThetaI / gridLen / sinThetaI;
	}
	else
	{
		A[index] = 0.0;
		B[index] = 1.0;
		C[index] = 0.0;
	}
}

void KaminoSolver::precomputeABCCoef()
{
	dim3 gridLayout = dim3(nPhi);
	dim3 blockLayout = dim3(nTheta);
	precomputeABCKernel<<<gridLayout, blockLayout>>>
	(this->gpuA, this->gpuB, this->gpuC, gridLen, nPhi, nTheta);
	checkCudaErrors(hipDeviceSynchronize());
}

void KaminoSolver::stepForward(fReal timeStep)
{
	this->timeStep = timeStep;
	advection();
	//std::cout << "Advection completed" << std::endl;
	geometric();
	//std::cout << "Geometric completed" << std::endl;
	bodyForce();
	//std::cout << "Body force application completed" << std::endl;
	projection();
	//std::cout << "Projection completed" << std::endl;
	this->timeElapsed += timeStep;
}

// Phi: 0 - 2pi  Theta: 0 - pi
bool validatePhiTheta(fReal & phi, fReal & theta)
{
	int loops = static_cast<int>(std::floor(theta / M_2PI));
	theta = theta - loops * M_2PI;
	// Now theta is in 0-2pi range

	bool isFlipped = false;

	if (theta > M_PI)
	{
		theta = M_2PI - theta;
		phi += M_PI;
		isFlipped = true;
	}

	loops = static_cast<int>(std::floor(phi / M_2PI));
	phi = phi - loops * M_2PI;
	// Now phi is in 0-2pi range

	return isFlipped;
}

void KaminoSolver::bodyForce()
{
	/// This is just a place holder now...
}

/* Tri-diagonal matrix solver */
void KaminoSolver::TDMSolve(fReal* a, fReal* b, fReal* c, fReal* d)
{
	// |b0 c0 0 ||x0| |d0|
	// |a1 b1 c1||x1|=|d1|
	// |0  a2 b2||x2| |d2|

	int n = nTheta;
	n--; // since we index from 0
	c[0] /= b[0];
	d[0] /= b[0];

	for (int i = 1; i < n; i++) {
		c[i] /= b[i] - a[i] * c[i - 1];
		d[i] = (d[i] - a[i] * d[i - 1]) / (b[i] - a[i] * c[i - 1]);
	}

	d[n] = (d[n] - a[n] * d[n - 1]) / (b[n] - a[n] * c[n - 1]);

	for (int i = n; i-- > 0;) {
		d[i] -= c[i] * d[i + 1];
	}
}

gridType KaminoSolver::getGridTypeAt(size_t x, size_t y)
{
	return this->cpuGridTypesBuffer[getIndex(x, y)];
}

KaminoQuantity* KaminoSolver::getAttributeNamed(std::string name)
{
	return (*this)[name];
}

void KaminoSolver::swapAttrBuffers()
{
	this->velPhi->swapGPUBuffer();
	this->velTheta->swapGPUBuffer();
}

void KaminoSolver::copyVelocityBack2CPU()
{
	this->velPhi->copyBackToCPU();
	this->velTheta->copyBackToCPU();
}


// <<<<<<<<<<
// OUTPUT >>>>>>>>>>


void KaminoSolver::write_data_bgeo(const std::string& s, const int frame)
{
	std::string file = s + std::to_string(frame) + ".bgeo";
	std::cout << "Writing to: " << file << std::endl;

	Partio::ParticlesDataMutable* parts = Partio::create();
	Partio::ParticleAttribute pH, vH, psH, dens;
	pH = parts->addAttribute("position", Partio::VECTOR, 3);
	vH = parts->addAttribute("v", Partio::VECTOR, 3);
	psH = parts->addAttribute("pressure", Partio::VECTOR, 1);
	dens = parts->addAttribute("density", Partio::VECTOR, 1);

	Eigen::Matrix<float, 3, 1> pos;
	Eigen::Matrix<float, 3, 1> vel;
	fReal pressure, densityValue;
	fReal velX, velY;

	KaminoQuantity* u = velPhi;
	KaminoQuantity* v = velTheta;
	fReal uRight, uLeft, vUp, vDown;

	size_t upi, vpi;

	for (size_t j = 0; j < nTheta; ++j) {
		for (size_t i = 0; i < nPhi; ++i) {
			uLeft = u->getCPUValueAt(i, j);
			i == (nPhi - 1) ? upi = 0 : upi = i + 1;
			vDown = v->getCPUValueAt(i, j);
			j == (nTheta - 1) ? vpi = 0 : vpi = j + 1;
			uRight = u->getCPUValueAt(upi, j);
			vUp = u->getCPUValueAt(i, vpi);

			velX = (uLeft + uRight) / 2.0;
			velY = (vUp + vDown) / 2.0;

			pos = Eigen::Matrix<float, 3, 1>(i * gridLen, j * gridLen, 0.0);
			vel = Eigen::Matrix<float, 3, 1>(0.0, velY, velX);
			mapVToSphere(pos, vel);
			mapPToSphere(pos);

			pressure = centeredAttr["p"]->getValueAt(i, j);
			densityValue = centeredAttr["density"]->getValueAt(i, j);

			int idx = parts->addParticle();
			float* p = parts->dataWrite<float>(pH, idx);
			float* v = parts->dataWrite<float>(vH, idx);
			float* ps = parts->dataWrite<float>(psH, idx);
			float* de = parts->dataWrite<float>(dens, idx);

			ps[0] = density * radius * pressure / timeStep;
			de[0] = densityValue;

			for (int k = 0; k < 3; ++k) {
				p[k] = pos(k, 0);
				v[k] = vel(k, 0);
			}
		}
	}

	Partio::write(file.c_str(), *parts);
	parts->release();
}

void KaminoSolver::mapPToSphere(Eigen::Matrix<float, 3, 1>& pos) const
{
	float theta = pos[1];
	float phi = pos[0];
	pos[0] = radius * sin(theta) * cos(phi);
	pos[2] = radius * sin(theta) * sin(phi);
	pos[1] = radius * cos(theta);
}

void KaminoSolver::mapVToSphere(Eigen::Matrix<float, 3, 1>& pos, Eigen::Matrix<float, 3, 1>& vel) const
{
	float theta = pos[1];
	float phi = pos[0];

	float u_theta = vel[1];
	float u_phi = vel[2];

	vel[0] = cos(theta) * cos(phi) * u_theta - sin(phi) * u_phi;
	vel[2] = cos(theta) * sin(phi) * u_theta + cos(phi) * u_phi;
	vel[1] = -sin(theta) * u_theta;
}

gridType* KaminoSolver::getGridTypeHandle()
{
	return this->cpuGridTypesBuffer;
}