#include "hip/hip_runtime.h"
# include "../include/KaminoSolver.h"
# include "../include/CubicSolver.h"

// CONSTRUCTOR / DESTRUCTOR >>>>>>>>>>

const int fftRank = 1;

KaminoSolver::KaminoSolver(size_t nPhi, size_t nTheta, fReal radius, fReal frameDuration,
	fReal A, int B, int C, int D, int E) :
	nPhi(nPhi), nTheta(nTheta), radius(radius), gridLen(M_2PI / nPhi), invGridLen(1.0 / gridLen), frameDuration(frameDuration),
	timeStep(0.0), timeElapsed(0.0),
	A(A), B(B), C(C), D(D), E(E)
{
	/// Replace it later with functions from hip/hip_runtime_api.h!
	checkCudaErrors(hipSetDevice(0));

	checkCudaErrors(hipMalloc((void **)&gpuUFourier,
		sizeof(ComplexFourier) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuUReal,
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuUImag,
		sizeof(fReal) * nPhi * nTheta));

	checkCudaErrors(hipMalloc((void **)&gpuFFourier,
		sizeof(ComplexFourier) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuFReal,
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)&gpuFImag,
		sizeof(fReal) * nPhi * nTheta));

	checkCudaErrors(hipMalloc((void **)(&gpuA),
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)(&gpuB),
		sizeof(fReal) * nPhi * nTheta));
	checkCudaErrors(hipMalloc((void **)(&gpuC),
		sizeof(fReal) * nPhi * nTheta));
	precomputeABCCoef();

	this->velPhi = new KaminoQuantity("velPhi", nPhi, nTheta,
		vPhiPhiOffset, vPhiThetaOffset);
	this->velTheta = new KaminoQuantity("velTheta", nPhi, nTheta - 1,
		vThetaPhiOffset, vThetaThetaOffset);
	this->pressure = new KaminoQuantity("p", nPhi, nTheta,
		centeredPhiOffset, centeredThetaOffset);

	/*this->cpuGridTypesBuffer = new gridType[nPhi * nTheta];
	checkCudaErrors(hipMalloc((void **)(this->gpuGridTypes),
		sizeof(gridType) * nPhi * nTheta));*/

	initialize_velocity();
	copyVelocity2GPU();

	initialize_boundary();
	//copyGridType2GPU();

	int sigLenArr[1];
	sigLenArr[0] = nPhi;
	checkCudaErrors((hipError_t)hipfftPlanMany(&kaminoPlan, fftRank, sigLenArr,
		NULL, 1, nPhi,
		NULL, 1, nPhi,
		HIPFFT_C2C, nTheta));
}

KaminoSolver::~KaminoSolver()
{
	checkCudaErrors(hipFree(gpuUFourier));
	checkCudaErrors(hipFree(gpuUReal));
	checkCudaErrors(hipFree(gpuUImag));

	checkCudaErrors(hipFree(gpuFFourier));
	checkCudaErrors(hipFree(gpuFReal));
	checkCudaErrors(hipFree(gpuFImag));
	
	checkCudaErrors(hipFree(gpuA));
	checkCudaErrors(hipFree(gpuB));
	checkCudaErrors(hipFree(gpuC));

	delete this->velPhi;
	delete this->velTheta;
	delete this->pressure;

	//delete[] cpuGridTypesBuffer;
	//checkCudaErrors(hipFree(gpuGridTypes));
}

void KaminoSolver::setTextureParams(table2D* tex)
{
	tex->addressMode[0] = hipAddressModeWrap;
	tex->addressMode[1] = hipAddressModeMirror;
	tex->filterMode = hipFilterModeLinear;
	tex->normalized = true;
}

void KaminoSolver::copyVelocity2GPU()
{
	velPhi->copyToGPU();
	velTheta->copyToGPU();
}

__global__ void precomputeABCKernel
(fReal* A, fReal* B, fReal* C, fReal gridLen, int nPhi, int nTheta)
{
	int nIndex = blockIdx.x;
	int n = nIndex - nPhi / 2;
	int i = threadIdx.x;
	int index = nIndex * nTheta + i;
	fReal thetaI = (i + centeredThetaOffset) * gridLen;

	fReal cosThetaI = cosf(thetaI);
	fReal sinThetaI = sinf(thetaI);

	if (n != 0)
	{
		A[index] = 1.0 / (gridLen * gridLen)
			- 0.5 * cosThetaI / gridLen / sinThetaI;
		B[index] = -2.0 / (gridLen * gridLen) - n * n / (sinThetaI * sinThetaI);
		C[index] = 1.0 / (gridLen * gridLen) + 0.5 * cosThetaI / gridLen / sinThetaI;
	}
	else
	{
		A[index] = 0.0;
		B[index] = 1.0;
		C[index] = 0.0;
	}
}

void KaminoSolver::precomputeABCCoef()
{
	dim3 gridLayout = dim3(nPhi);
	dim3 blockLayout = dim3(nTheta);
	precomputeABCKernel<<<gridLayout, blockLayout>>>
	(this->gpuA, this->gpuB, this->gpuC, gridLen, nPhi, nTheta);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void KaminoSolver::stepForward(fReal timeStep)
{
	this->timeStep = timeStep;
	advection();
	std::cout << "Advection completed" << std::endl;
	//geometric();
	//std::cout << "Geometric completed" << std::endl;
	//bodyForce();
	//std::cout << "Body force application completed" << std::endl;
	//projection();
	//std::cout << "Projection completed" << std::endl;
	this->timeElapsed += timeStep;

	velPhi->copyBackToCPU();
	velTheta->copyBackToCPU();
}

// Phi: 0 - 2pi  Theta: 0 - pi
bool validatePhiTheta(fReal & phi, fReal & theta)
{
	int loops = static_cast<int>(std::floor(theta / M_2PI));
	theta = theta - loops * M_2PI;
	// Now theta is in 0-2pi range

	bool isFlipped = false;

	if (theta > M_PI)
	{
		theta = M_2PI - theta;
		phi += M_PI;
		isFlipped = true;
	}

	loops = static_cast<int>(std::floor(phi / M_2PI));
	phi = phi - loops * M_2PI;
	// Now phi is in 0-2pi range

	return isFlipped;
}

void KaminoSolver::bodyForce()
{
	/// This is just a place holder now...
}

/* Tri-diagonal matrix solver */
void KaminoSolver::TDMSolve(fReal* a, fReal* b, fReal* c, fReal* d)
{
	// |b0 c0 0 ||x0| |d0|
	// |a1 b1 c1||x1|=|d1|
	// |0  a2 b2||x2| |d2|

	int n = nTheta;
	n--; // since we index from 0
	c[0] /= b[0];
	d[0] /= b[0];

	for (int i = 1; i < n; i++) {
		c[i] /= b[i] - a[i] * c[i - 1];
		d[i] = (d[i] - a[i] * d[i - 1]) / (b[i] - a[i] * c[i - 1]);
	}

	d[n] = (d[n] - a[n] * d[n - 1]) / (b[n] - a[n] * c[n - 1]);

	for (int i = n; i-- > 0;) {
		d[i] -= c[i] * d[i + 1];
	}
}

gridType KaminoSolver::getGridTypeAt(size_t x, size_t y)
{
	return this->cpuGridTypesBuffer[getIndex(x, y)];
}

/*KaminoQuantity* KaminoSolver::getAttributeNamed(std::string name)
{
	return (*this)[name];
}*/

void KaminoSolver::swapAttrBuffers()
{
	this->velPhi->swapGPUBuffer();
	this->velTheta->swapGPUBuffer();
}

void KaminoSolver::copyVelocityBack2CPU()
{
	this->velPhi->copyBackToCPU();
	this->velTheta->copyBackToCPU();
}


// <<<<<<<<<<
// OUTPUT >>>>>>>>>>


void KaminoSolver::write_data_bgeo(const std::string& s, const int frame)
{
	std::string file = s + std::to_string(frame) + ".bgeo";
	std::cout << "Writing to: " << file << std::endl;

	Partio::ParticlesDataMutable* parts = Partio::create();
	Partio::ParticleAttribute pH, vH;// , psH, dens;
	pH = parts->addAttribute("position", Partio::VECTOR, 3);
	vH = parts->addAttribute("v", Partio::VECTOR, 3);

	vec3 pos;
	vec3 vel;

	size_t iWest, iEast, jNorth, jSouth;
	fReal uWest, uEast, vNorth, vSouth;

	for (size_t j = 0; j < nTheta; ++j)
	{
		for (size_t i = 0; i < nPhi; ++i)
		{
			iWest = i;
			uWest = velPhi->getCPUValueAt(iWest, j);
			i == (nPhi - 1) ? iEast = 0 : iEast = i + 1;
			uEast = velPhi->getCPUValueAt(iEast, j);

			if (j == 0)
			{
				jNorth = jSouth = 0;
			}
			else if (j == nTheta - 1)
			{
				jNorth = jSouth = nTheta - 2;
			}
			else
			{
				jNorth = j - 1;
				jSouth = j;
			}
			vNorth = velTheta->getCPUValueAt(i, jNorth);
			vSouth = velTheta->getCPUValueAt(i, jSouth);

			fReal velocityPhi, velocityTheta;
			velocityPhi = (uWest + uEast) / 2.0;
			velocityTheta = (vNorth + vSouth) / 2.0;

			pos = vec3((i + centeredPhiOffset) * gridLen, (j + centeredThetaOffset) * gridLen, 0.0);
			vel = vec3(0.0, velocityTheta, velocityPhi);
			mapVToSphere(pos, vel);
			mapPToSphere(pos);

			int idx = parts->addParticle();
			float* p = parts->dataWrite<float>(pH, idx);
			float* v = parts->dataWrite<float>(vH, idx);
			
			for (int k = 0; k < 3; ++k) 
			{
				p[k] = pos[k];
				v[k] = vel[k];
			}
		}
	}

	Partio::write(file.c_str(), *parts);
	parts->release();
}

void KaminoSolver::mapPToSphere(vec3& pos) const
{
	float theta = pos[1];
	float phi = pos[0];
	pos[0] = radius * sin(theta) * cos(phi);
	pos[2] = radius * sin(theta) * sin(phi);
	pos[1] = radius * cos(theta);
}

void KaminoSolver::mapVToSphere(vec3& pos, vec3& vel) const
{
	float theta = pos[1];
	float phi = pos[0];

	float u_theta = vel[1];
	float u_phi = vel[2];

	vel[0] = cos(theta) * cos(phi) * u_theta - sin(phi) * u_phi;
	vel[2] = cos(theta) * sin(phi) * u_theta + cos(phi) * u_phi;
	vel[1] = -sin(theta) * u_theta;
}
