#include "hip/hip_runtime.h"
# include "../include/KaminoSolver.h"

__global__ void crKernel(fReal *d_a, fReal *d_b, fReal *d_c, fReal *d_d, fReal *d_x);

__global__ void fillDivergenceKernel
(ComplexFourier* outputF,
	size_t nTheta, size_t nPhi,
	fReal gridLen, fReal radius, fReal timeStep)
{
	int gridPhiId = threadIdx.x;
	int gridThetaId = blockIdx.x;
	fReal gridPhiCoord = ((fReal)gridPhiId + centeredPhiOffset) * gridLen;
	fReal gridThetaCoord = ((fReal)gridThetaId + centeredThetaOffset) * gridLen;

	fReal uEast = 0.0;
	fReal uWest = 0.0;
	fReal vNorth = 0.0;
	fReal vSouth = 0.0;

	fReal halfStep = 0.5 * gridLen;
	fReal phiEast = gridPhiCoord + halfStep;
	fReal phiWest = gridPhiCoord - halfStep;
	fReal thetaNorth = gridThetaCoord - halfStep;
	fReal thetaSouth = gridThetaCoord + halfStep;

	// sample the vPhi at gridThetaCoord
	fReal thetaTex = (gridThetaCoord - vPhiThetaOffset * gridLen) / vPhiThetaNorm;
	// sample the vTheta at gridPhiCoord
	fReal phiTex = (gridPhiCoord - vThetaPhiOffset * gridLen) / vThetaPhiNorm;

	fReal phiEastTex = (phiEast - vPhiPhiOffset * gridLen) / vPhiPhiNorm;
	fReal phiWestTex = (phiWest - vPhiPhiOffset * gridLen) / vPhiPhiNorm;

	uEast = tex2D(texVelPhi, phiEastTex, thetaTex);
	uWest = tex2D(texVelPhi, phiWestTex, thetaTex);

	if (gridThetaId != 0)
	{
		fReal thetaNorthTex = (thetaNorth - vThetaThetaOffset * gridLen) / vThetaThetaNorm;
		vNorth = tex2D(texVelTheta, phiTex, thetaNorthTex);
	}
	if (gridThetaId != nTheta - 1)
	{
		fReal thetaSouthTex = (thetaSouth - vThetaThetaOffset * gridLen) / vThetaThetaNorm;
		vSouth = tex2D(texVelTheta, phiTex, thetaSouthTex);
	}

	fReal invGridSine = 1.0 / sinf(gridThetaCoord);
	fReal sinNorth = sinf(thetaNorth);
	fReal sinSouth = sinf(thetaSouth);
	fReal factor = invGridSine / gridLen;
	fReal termTheta = factor * (vSouth * sinSouth - vNorth * sinNorth);
	fReal termPhi = factor * (uEast - uWest);

	fReal div = termTheta + termPhi;

	ComplexFourier f;
	f.x = div;
	f.y = 0.0;
	outputF[gridThetaId * nPhi + gridPhiId] = f;
}

__global__ void shiftFKernel
(ComplexFourier* FFourierInput, fReal* FFourierShiftedReal, fReal* FFourierShiftedImag,
	size_t nTheta, size_t nPhi)
{
	int nIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	int fftIndex = nPhi / 2 - nIdx;
	if (fftIndex < 0)
		fftIndex += nPhi;
	//FFourierShifted[thetaIdx * nPhi + phiIdx] = FFourierInput[thetaIdx * nPhi + fftIndex];
	FFourierShiftedReal[nIdx * nTheta + thetaIdx] = FFourierInput[thetaIdx * nPhi + fftIndex].x;
	FFourierShiftedImag[nIdx * nTheta + thetaIdx] = FFourierInput[thetaIdx * nPhi + fftIndex].y;
}

__global__ void copy2UFourier
(ComplexFourier* UFourierOutput, fReal* UFourierReal, fReal* UFourierImag,
	size_t nTheta, size_t nPhi)
{
	int nIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	UFourierOutput[thetaIdx * nPhi + nIdx].x = UFourierReal[nIdx * nTheta + thetaIdx];
	UFourierOutput[thetaIdx * nPhi + nIdx].y = UFourierImag[nIdx * nTheta + thetaIdx];
}

__global__ void shiftUKernel
(ComplexFourier* UFourierInput, fReal* pressure,
	size_t nTheta, size_t nPhi, size_t nPressurePitch)
{
	int phiIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	int fftIndex = 0;
	fReal zeroComponent = UFourierInput[thetaIdx * nPhi + nPhi / 2].x;
	if (phiIdx != 0)
		fftIndex = nPhi - phiIdx;
	fReal pressureVal = 0.0;
	int bit = 0;
	if (phiIdx & 2 == 0)
		bit = 1;
	else
		bit = -1;
	pressureVal = bit * UFourierInput[thetaIdx * nPhi + phiIdx].x - zeroComponent;
	pressure[thetaIdx * nPressurePitch + phiIdx] = pressureVal;
}

__global__ void applyPressureTheta
(fReal* output,
	size_t nTheta, size_t nPhi,
	fReal gridLen)
{
	int thetaId = threadIdx.x;
	int phiId = blockIdx.x;

	fReal gPhi = ((fReal)phiId + vThetaPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vThetaThetaOffset) * gridLen;
	fReal thetaSouth = gTheta + 0.5 * gridLen;
	fReal thetaNorth = gTheta - 0.5 * gridLen;

	fReal texPhi = (fReal)phiId / nPhi;
	fReal texTheta = (fReal)thetaId / nTheta;
	fReal texThetaNorth = (thetaNorth - vThetaThetaOffset * gridLen) / pressureThetaNorm;
	fReal texThetaSouth = (thetaSouth - vThetaThetaOffset * gridLen) / pressureThetaNorm;

	fReal previousVTheta = tex2D(texVelTheta, texPhi, texTheta);
	fReal pressureNorth = tex2D(texPressure, texPhi, texThetaNorth);
	fReal pressureSouth = tex2D(texPressure, texPhi, texThetaSouth);

	fReal pressureTheta = pressureSouth - pressureNorth;
	fReal deltaVTheta = -pressureTheta / gridLen;

	output[thetaId * nPhi + phiId] = previousVTheta + deltaVTheta;
}
__global__ void applyPressurePhi
(fReal* output,
	size_t nTheta, size_t nPhi,
	fReal gridLen)
{
	int thetaId = threadIdx.x;
	int phiId = blockIdx.x;

	fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vPhiThetaOffset) * gridLen;
	fReal phiEast = gPhi + 0.5 * gridLen;
	fReal phiWest = gPhi - 0.5 * gridLen;

	fReal texPhi = (fReal)phiId / nPhi;
	fReal texTheta = (fReal)thetaId / nTheta;
	fReal texPhiEast = (phiEast - vPhiPhiOffset * gridLen) / pressurePhiNorm;
	fReal texPhiWest = (phiWest - vPhiPhiOffset * gridLen) / pressurePhiNorm;

	fReal previousVPhi = tex2D(texVelPhi, texPhi, texTheta);
	fReal pressureEast = tex2D(texPressure, texPhiEast, texTheta);
	fReal pressureWest = tex2D(texPressure, texPhiWest, texTheta);

	fReal pressurePhi = pressureEast - pressureWest;
	fReal deltaVPhi = -pressurePhi / (gridLen * sinf(gTheta));

	output[thetaId * nPhi + phiId] = previousVPhi + deltaVPhi;
}

void KaminoSolver::projection()
{
	velPhi->bindTexture(texVelPhi);
	velTheta->bindTexture(texVelTheta);

	dim3 gridLayout(nTheta);
	dim3 blockLayout(nPhi);
	fillDivergenceKernel<<<gridLayout, blockLayout>>>
	(gpuFFourier, 
		nTheta, nPhi,
		gridLen, radius, timeStep);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
		gpuFFourier, gpuFFourier, HIPFFT_BACKWARD));
	checkCudaErrors(hipGetLastError());



	shiftFKernel<<<gridLayout, blockLayout>>>
	(gpuFFourier, gpuFReal, gpuFImag, nTheta, nPhi);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// Now gpuFDivergence stores all the Fn



	gridLayout = dim3(nPhi);
	blockLayout = dim3(nTheta / 2);
	const unsigned sharedMemSize = nTheta * 5 * sizeof(fReal);
	crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFReal, this->gpuUReal);
	checkCudaErrors(hipGetLastError());
	crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFImag, this->gpuUImag);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	gridLayout = dim3(nTheta);
	blockLayout = dim3(nPhi);
	copy2UFourier<<<gridLayout, blockLayout>>>
	(this->gpuUFourier, this->gpuUReal, this->gpuUImag, nTheta, nPhi);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
		gpuUFourier, gpuUFourier, HIPFFT_FORWARD));
	checkCudaErrors(hipGetLastError());



	shiftUKernel<<<gridLayout, blockLayout>>>
	(gpuUFourier, pressure->getGPUThisStep(),
		nTheta, nPhi, pressure->getThisStepPitch());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	velPhi->bindTexture(texVelPhi);
	velTheta->bindTexture(texVelTheta);
	pressure->bindTexture(texPressure);

	gridLayout = dim3(velTheta->getNTheta());
	blockLayout = dim3(velTheta->getNPhi());
	applyPressureTheta<<<gridLayout, blockLayout>>>
	(velTheta->getGPUNextStep(),
		velTheta->getNTheta(), velTheta->getNPhi(), gridLen);
	checkCudaErrors(hipGetLastError());

	gridLayout = dim3(velPhi->getNTheta());
	blockLayout = dim3(velPhi->getNPhi());
	applyPressurePhi<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(),
		velPhi->getNTheta(), velPhi->getNPhi(), gridLen);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipDeviceSynchronize());
}