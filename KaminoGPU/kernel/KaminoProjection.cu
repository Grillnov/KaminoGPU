#include "hip/hip_runtime.h"
# include "../include/KaminoSolver.cuh"

static table2D texProjVelPhi;
static table2D texProjVelTheta;
static table2D texProjPressure;

static __constant__ size_t nPhiGlobal;
static __constant__ size_t nThetaGlobal;
static __constant__ fReal radiusGlobal;
static __constant__ fReal timeStepGlobal;
static __constant__ fReal gridLenGlobal;

__global__ void crKernel(fReal *d_a, fReal *d_b, fReal *d_c, fReal *d_d, fReal *d_x);

__global__ void fillDivergenceKernel
(ComplexFourier* outputF, fReal* velPhi, fReal* velTheta,
	size_t velPhiPitchInElements, size_t velThetaPitchInElements)
{
	int gridPhiId = threadIdx.x;
	int gridThetaId = blockIdx.x;
	//fReal gridPhiCoord = ((fReal)gridPhiId + centeredPhiOffset) * gridLen;
	fReal gridThetaCoord = ((fReal)gridThetaId + centeredThetaOffset) * gridLenGlobal;

	fReal uEast = 0.0;
	fReal uWest = 0.0;
	fReal vNorth = 0.0;
	fReal vSouth = 0.0;

	fReal halfStep = 0.5 * gridLenGlobal;
	
	fReal thetaSouth = gridThetaCoord + halfStep;
	fReal thetaNorth = gridThetaCoord - halfStep;

	int phiIdWest = gridPhiId;
	int phiIdEast = (phiIdWest + 1) % nPhiGlobal;

	uWest = velPhi[gridThetaId * velPhiPitchInElements + phiIdWest];
	uEast = velPhi[gridThetaId * velPhiPitchInElements + phiIdEast];

	if (gridThetaId != 0)
	{
		int thetaNorthIdx = gridThetaId - 1;
		vNorth = velTheta[thetaNorthIdx * velThetaPitchInElements + gridPhiId];
	}
	if (gridThetaId != nThetaGlobal - 1)
	{
		int thetaSouthIdx = gridThetaId;
		vSouth = velTheta[thetaSouthIdx * velThetaPitchInElements + gridPhiId];
	}

	fReal invGridSine = 1.0 / sinf(gridThetaCoord);
	fReal sinNorth = sinf(thetaNorth);
	fReal sinSouth = sinf(thetaSouth);
	fReal factor = invGridSine / gridLenGlobal;
	fReal termTheta = factor * (vSouth * sinSouth - vNorth * sinNorth);
	fReal termPhi = factor * (uEast - uWest);

	fReal div = termTheta + termPhi;

	ComplexFourier f;
	f.x = div;
	f.y = 0.0;
	outputF[gridThetaId * nPhiGlobal + gridPhiId] = f;
}

__global__ void shiftFKernel
(ComplexFourier* FFourierInput, fReal* FFourierShiftedReal, fReal* FFourierShiftedImag)
{
	int nIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	int fftIndex = nPhiGlobal / 2 - nIdx;
	if (fftIndex < 0)
		fftIndex += nPhiGlobal;
	//FFourierShifted[thetaIdx * nPhi + phiIdx] = FFourierInput[thetaIdx * nPhi + fftIndex];
	fReal real = FFourierInput[thetaIdx * nPhiGlobal + fftIndex].x / (fReal)nPhiGlobal;
	fReal imag = FFourierInput[thetaIdx * nPhiGlobal + fftIndex].y / (fReal)nPhiGlobal;
	FFourierShiftedReal[nIdx * nThetaGlobal + thetaIdx] = real;
	FFourierShiftedImag[nIdx * nThetaGlobal + thetaIdx] = imag;
}

__global__ void copy2UFourier
(ComplexFourier* UFourierOutput, fReal* UFourierReal, fReal* UFourierImag)
{
	int nIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	ComplexFourier u;
	u.x = UFourierReal[nIdx * nThetaGlobal + thetaIdx];
	u.y = UFourierImag[nIdx * nThetaGlobal + thetaIdx];
	UFourierOutput[thetaIdx * nPhiGlobal + nIdx] = u;
}

__global__ void cacheZeroComponents
(fReal* zeroComponentCache, ComplexFourier* input)
{
	int thetaIdx = threadIdx.x;
	zeroComponentCache[thetaIdx] = input[thetaIdx * nPhiGlobal + nPhiGlobal / 2].x;
}

__global__ void shiftUKernel
(ComplexFourier* UFourierInput, fReal* pressure, fReal* zeroComponentCache,
	size_t nPressurePitchInElements)
{
	int phiIdx = threadIdx.x;
	int thetaIdx = blockIdx.x;
	int fftIndex = 0;
	fReal zeroComponent = zeroComponentCache[thetaIdx];
	if (phiIdx != 0)
		fftIndex = nPhiGlobal - phiIdx;
	fReal pressureVal;

	if (phiIdx % 2 == 0)
		pressureVal = UFourierInput[thetaIdx * nPhiGlobal + fftIndex].x - zeroComponent;
	else
		pressureVal = -UFourierInput[thetaIdx * nPhiGlobal + fftIndex].x - zeroComponent;
	
	pressure[thetaIdx * nPressurePitchInElements + phiIdx] = pressureVal;
}

__global__ void applyPressureTheta
(fReal* output, fReal* prev, fReal* pressure,
	size_t nPitchInElementsPressure, size_t nPitchInElementsVTheta)
{
	int phiId = threadIdx.x;
	int thetaId = blockIdx.x;

	int pressureThetaNorthId = thetaId;
	int pressureThetaSouthId = thetaId + 1;
	fReal pressureNorth = pressure[pressureThetaNorthId * nPitchInElementsPressure + phiId];
	fReal pressureSouth = pressure[pressureThetaSouthId * nPitchInElementsPressure + phiId];

	fReal deltaVTheta = (pressureSouth - pressureNorth) / (-gridLenGlobal);
	fReal previousVTheta = prev[thetaId * nPitchInElementsVTheta + phiId];
	output[thetaId * nPitchInElementsVTheta + phiId] = previousVTheta + deltaVTheta;
}
__global__ void applyPressurePhi
(fReal* output, fReal* prev, fReal* pressure,
	size_t nPitchInElementsPressure, size_t nPitchInElementsVPhi)
{
	int phiId = threadIdx.x;
	int thetaId = blockIdx.x;

	int pressurePhiWestId;
	if (phiId == 0)
		pressurePhiWestId = nPhiGlobal - 1;
	else
		pressurePhiWestId = phiId - 1;
	int pressurePhiEastId = phiId;

	fReal pressureWest = pressure[thetaId * nPitchInElementsPressure + pressurePhiWestId];
	fReal pressureEast = pressure[thetaId * nPitchInElementsPressure + pressurePhiEastId];

	fReal thetaBelt = (thetaId + centeredThetaOffset) * gridLenGlobal;
	fReal deltaVPhi = (pressureEast - pressureWest) / (-gridLenGlobal * sinf(thetaBelt));
	fReal previousVPhi = prev[thetaId * nPitchInElementsVPhi + phiId];
	output[thetaId * nPitchInElementsVPhi + phiId] = previousVPhi + deltaVPhi;
}

void KaminoSolver::projection()
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nPhiGlobal), &(this->nPhi), sizeof(size_t)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nThetaGlobal), &(this->nTheta), sizeof(size_t)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(radiusGlobal), &(this->radius), sizeof(fReal)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(timeStepGlobal), &(this->timeStep), sizeof(fReal)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gridLenGlobal), &(this->gridLen), sizeof(fReal)));



	dim3 gridLayout(nTheta);
	dim3 blockLayout(nPhi);
	fillDivergenceKernel<<<gridLayout, blockLayout>>>
	(this->gpuFFourier, this->velPhi->getGPUThisStep(), this->velTheta->getGPUThisStep(),
		this->velPhi->getThisStepPitchInElements(), this->velTheta->getThisStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	// Note that cuFFT inverse returns results are SigLen times larger
	checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
		this->gpuFFourier, this->gpuFFourier, HIPFFT_BACKWARD));
	checkCudaErrors(hipGetLastError());



	// Siglen is nPhi
	gridLayout = dim3(nTheta);
	blockLayout = dim3(nPhi);
	shiftFKernel<<<gridLayout, blockLayout>>>
	(gpuFFourier, gpuFReal, gpuFImag);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// Now gpuFDivergence stores all the Fn



	gridLayout = dim3(nPhi);
	blockLayout = dim3(nTheta / 2);
	const unsigned sharedMemSize = nTheta * 5 * sizeof(fReal);
	crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFReal, this->gpuUReal);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	gridLayout = dim3(nPhi);
	blockLayout = dim3(nTheta / 2);
	crKernel<<<gridLayout, blockLayout, sharedMemSize>>>
	(this->gpuA, this->gpuB, this->gpuC, this->gpuFImag, this->gpuUImag);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	gridLayout = dim3(nTheta);
	blockLayout = dim3(nPhi);
	copy2UFourier<<<gridLayout, blockLayout>>>
	(this->gpuUFourier, this->gpuUReal, this->gpuUImag);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	gridLayout = dim3(1);
	blockLayout = dim3(nTheta);
	cacheZeroComponents<<<gridLayout, blockLayout>>>
	(gpuFZeroComponent, gpuUFourier);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	checkCudaErrors((hipError_t)hipfftExecC2C(this->kaminoPlan,
		this->gpuUFourier, this->gpuUFourier, HIPFFT_FORWARD));
	checkCudaErrors(hipGetLastError());



	gridLayout = dim3(nTheta);
	blockLayout = dim3(nPhi);
	shiftUKernel<<<gridLayout, blockLayout>>>
	(gpuUFourier, pressure->getGPUThisStep(), this->gpuFZeroComponent,
		pressure->getThisStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//pressure->copyBackToCPU();

	gridLayout = dim3(velTheta->getNTheta());
	blockLayout = dim3(velTheta->getNPhi());
	applyPressureTheta<<<gridLayout, blockLayout>>>
		(velTheta->getGPUNextStep(), velTheta->getGPUThisStep(), pressure->getGPUThisStep(),
			pressure->getThisStepPitchInElements(), velTheta->getNextStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	gridLayout = dim3(velPhi->getNTheta());
	blockLayout = dim3(velPhi->getNPhi());
	applyPressurePhi<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(), velPhi->getGPUThisStep(), pressure->getGPUThisStep(),
		pressure->getThisStepPitchInElements(), velPhi->getNextStepPitchInElements());
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	swapAttrBuffers();
}