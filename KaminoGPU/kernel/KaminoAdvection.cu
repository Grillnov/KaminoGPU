#include "hip/hip_runtime.h"
# include "KaminoSolver.h"

static table2D texAdvVelPhi;
static table2D texAdvVelTheta;

__global__ void advectionVPhiKernel
	(fReal* attributeOutput,
	size_t nTheta, size_t nPhi, size_t nPitchInElements,
	fReal gridLen, fReal radius, fReal timeStep)
{
	// Index
    int phiId = threadIdx.x;
	int thetaId = blockIdx.x;
	// Coord in phi-theta space
	fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vPhiThetaOffset) * gridLen;
	
	// Coord in u-v texture space
	fReal gPhiTex = (fReal)phiId / nPhi;
	fReal gThetaTex = (fReal)thetaId / nTheta;

	// Sample the speed
	fReal guPhi = tex2D<fReal>(texAdvVelPhi, gPhiTex, gThetaTex);
	fReal guTheta = tex2D<fReal>(texAdvVelTheta, gPhiTex, gThetaTex);

	fReal latRadius = radius * sinf(gTheta);
	fReal cofPhi = timeStep / latRadius;
	fReal cofTheta = timeStep / radius;

	fReal deltaPhi = guPhi * cofPhi;
	fReal deltaTheta = guTheta * cofTheta;

	// Traced halfway in phi-theta space
	fReal midPhi = gPhi - 0.5 * deltaPhi;
	fReal midTheta = gTheta - 0.5 * deltaTheta;
	fReal midPhiTex = (midPhi - vPhiPhiOffset * gridLen) / vPhiPhiNorm;
	fReal midThetaTex = (midTheta - vPhiThetaOffset * gridLen) / vPhiThetaNorm;

	fReal muPhi = tex2D<fReal>(texAdvVelPhi, midPhiTex, midThetaTex);
	fReal muTheta = tex2D<fReal>(texAdvVelTheta, midPhiTex, midThetaTex);

	fReal averuPhi = 0.5 * (muPhi + guPhi);
	fReal averuTheta = 0.5 * (muTheta + guTheta);

	deltaPhi = averuPhi * cofPhi;
	deltaTheta = averuTheta * cofTheta;
	fReal pPhi = gPhi - deltaPhi;
	fReal pTheta = gTheta - deltaTheta;
	fReal pPhiTex = (pPhi - vPhiPhiOffset * gridLen) / vPhiPhiNorm;
	fReal pThetaTex = (pTheta - vPhiThetaOffset * gridLen) / vPhiThetaNorm;

	fReal advectedVal = tex2D<fReal>(texAdvVelPhi, pPhiTex, pThetaTex);

	attributeOutput[thetaId * nPitchInElements + phiId] = advectedVal;
};

__global__ void advectionVThetaKernel
(fReal* attributeOutput,
	size_t nTheta, size_t nPhi, size_t nPitchInElements,
	fReal gridLen, fReal radius, fReal timeStep)
{
	// Index
	int phiId = threadIdx.x;
	int thetaId = blockIdx.x;
	// Coord in phi-theta space
	fReal gPhi = ((fReal)phiId + vThetaPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vThetaThetaOffset) * gridLen;

	// Coord in u-v texture space
	fReal gPhiTex = (fReal)phiId / nPhi;
	fReal gThetaTex = (fReal)thetaId / nTheta;

	// Sample the speed
	fReal guPhi = tex2D<fReal>(texAdvVelPhi, gPhiTex, gThetaTex);
	fReal guTheta = tex2D<fReal>(texAdvVelTheta, gPhiTex, gThetaTex);

	fReal latRadius = radius * sinf(gTheta);
	fReal cofPhi = timeStep / latRadius;
	fReal cofTheta = timeStep / radius;

	fReal deltaPhi = guPhi * cofPhi;
	fReal deltaTheta = guTheta * cofTheta;

	// Traced halfway in phi-theta space
	fReal midPhi = gPhi - 0.5 * deltaPhi;
	fReal midTheta = gTheta - 0.5 * deltaTheta;
	fReal midPhiTex = (midPhi - vThetaPhiOffset * gridLen) / vThetaPhiNorm;
	fReal midThetaTex = (midTheta - vThetaThetaOffset * gridLen) / vThetaThetaNorm;

	fReal muPhi = tex2D(texAdvVelPhi, midPhiTex, midThetaTex);
	fReal muTheta = tex2D(texAdvVelTheta, midPhiTex, midThetaTex);

	fReal averuPhi = 0.5 * (muPhi + guPhi);
	fReal averuTheta = 0.5 * (muTheta + guTheta);

	deltaPhi = averuPhi * cofPhi;
	deltaTheta = averuTheta * cofTheta;
	fReal pPhi = gPhi - deltaPhi;
	fReal pTheta = gTheta - deltaTheta;
	fReal pPhiTex = (pPhi - vThetaPhiOffset * gridLen) / vThetaPhiNorm;
	fReal pThetaTex = (pTheta - vThetaThetaOffset * gridLen) / vThetaThetaNorm;

	fReal advectedVal = tex2D(texAdvVelTheta, pPhiTex, pThetaTex);

	attributeOutput[thetaId * nPitchInElements + phiId] = advectedVal;
};

void KaminoSolver::advection()
{
	//bindVelocity2Tex(texVelPhi, texVelTheta);
	setTextureParams(&texAdvVelPhi);
	setTextureParams(&texAdvVelTheta);
	velPhi->bindTexture(&texAdvVelPhi);
	velTheta->bindTexture(&texAdvVelTheta);


	
	///kernel call goes here
	// Advect Phi
	dim3 gridLayout = dim3(velPhi->getNTheta());
	dim3 blockLayout = dim3(velPhi->getNPhi());
	advectionVPhiKernel<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(), velPhi->getNTheta(), velPhi->getNPhi(), velPhi->getNextStepPitch() / sizeof(fReal),
	gridLen, radius, timeStep);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Advect Theta
	
	gridLayout = dim3(velTheta->getNTheta());
	blockLayout = dim3(velTheta->getNPhi());
	advectionVThetaKernel<<<gridLayout, blockLayout>>>
	(velTheta->getGPUNextStep(), velTheta->getNTheta(), velTheta->getNPhi(), velTheta->getNextStepPitch() / sizeof(fReal),
	gridLen, radius, timeStep);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



	velPhi->unbindTexture(&texAdvVelPhi);
	velTheta->unbindTexture(&texAdvVelTheta);

	swapAttrBuffers();
}