#include "hip/hip_runtime.h"
# include "KaminoSolver.h"

__global__ void advectionVPhiKernel
	(fReal* attributeOutput,
	size_t nTheta, size_t nPhi, size_t nPitch,
	fReal gridLen, fReal radius, fReal timeStep)
{
	// Index
    int phiId = threadIdx.x;
	int thetaId = blockIdx.x;
	// Coord in phi-theta space
	fReal gPhi = ((fReal)phiId + vPhiPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vPhiThetaOffset) * gridLen;
	
	// Coord in u-v texture space
	fReal gPhiTex = (fReal)phiId / nPhi;
	fReal gThetaTex = (fReal)thetaId / nTheta;

	// Sample the speed
	fReal guPhi = tex2D(texVelPhi, gPhiTex, gThetaTex);
	fReal guTheta = tex2D(texVelTheta, gPhiTex, gThetaTex);

	fReal latRadius = radius * sinf(gTheta);
	fReal cofPhi = timeStep / latRadius;
	fReal cofTheta = timeStep / radius;

	fReal deltaPhi = guPhi * cofPhi;
	fReal deltaTheta = guTheta * cofTheta;

	// Traced halfway in phi-theta space
	fReal midPhi = gPhi - 0.5 * deltaPhi;
	fReal midTheta = gTheta - 0.5 * deltaTheta;
	fReal midPhiTex = (midPhi - vPhiPhiOffset * gridLen) / vPhiPhiNorm;
	fReal midThetaTex = (midTheta - vPhiThetaOffset * gridLen) / vPhiThetaNorm;

	fReal muPhi = tex2D(texVelPhi, midPhiTex, midThetaTex);
	fReal muTheta = tex2D(texVelTheta, midPhiTex, midThetaTex);

	fReal averuPhi = 0.5 * (muPhi + guPhi);
	fReal averuTheta = 0.5 * (muTheta + guTheta);

	deltaPhi = averuPhi * cofPhi;
	deltaTheta = averuTheta * cofTheta;
	fReal pPhi = gPhi - deltaPhi;
	fReal pTheta = gTheta - deltaTheta;
	fReal pPhiTex = (pPhi - vPhiPhiOffset * gridLen) / vPhiPhiNorm;
	fReal pThetaTex = (pTheta - vPhiThetaOffset * gridLen) / vPhiThetaNorm;

	fReal advectedVal = tex2D(texBeingAdvected, pPhiTex, pThetaTex);

	attributeOutput[thetaId * nPitch + phiId] = advectedVal;
};

__global__ void advectionVThetaKernel
(fReal* attributeOutput,
	size_t nTheta, size_t nPhi, size_t nPitch,
	fReal gridLen, fReal radius, fReal timeStep)
{
	// Index
	int phiId = threadIdx.x;
	int thetaId = blockIdx.x;
	// Coord in phi-theta space
	fReal gPhi = ((fReal)phiId + vThetaPhiOffset) * gridLen;
	fReal gTheta = ((fReal)thetaId + vThetaThetaOffset) * gridLen;

	// Coord in u-v texture space
	fReal gPhiTex = (fReal)phiId / nPhi;
	fReal gThetaTex = (fReal)thetaId / nTheta;

	// Sample the speed
	fReal guPhi = tex2D(texVelPhi, gPhiTex, gThetaTex);
	fReal guTheta = tex2D(texVelTheta, gPhiTex, gThetaTex);

	fReal latRadius = radius * sinf(gTheta);
	fReal cofPhi = timeStep / latRadius;
	fReal cofTheta = timeStep / radius;

	fReal deltaPhi = guPhi * cofPhi;
	fReal deltaTheta = guTheta * cofTheta;

	// Traced halfway in phi-theta space
	fReal midPhi = gPhi - 0.5 * deltaPhi;
	fReal midTheta = gTheta - 0.5 * deltaTheta;
	fReal midPhiTex = (midPhi - vThetaPhiOffset * gridLen) / vThetaPhiNorm;
	fReal midThetaTex = (midTheta - vThetaThetaOffset * gridLen) / vThetaThetaNorm;

	fReal muPhi = tex2D(texVelPhi, midPhiTex, midThetaTex);
	fReal muTheta = tex2D(texVelTheta, midPhiTex, midThetaTex);

	fReal averuPhi = 0.5 * (muPhi + guPhi);
	fReal averuTheta = 0.5 * (muTheta + guTheta);

	deltaPhi = averuPhi * cofPhi;
	deltaTheta = averuTheta * cofTheta;
	fReal pPhi = gPhi - deltaPhi;
	fReal pTheta = gTheta - deltaTheta;
	fReal pPhiTex = (pPhi - vThetaPhiOffset * gridLen) / vThetaPhiNorm;
	fReal pThetaTex = (pTheta - vThetaThetaOffset * gridLen) / vThetaThetaNorm;

	fReal advectedVal = tex2D(texBeingAdvected, pPhiTex, pThetaTex);

	attributeOutput[thetaId * nPitch + phiId] = advectedVal;
};

/*(fReal* attributeOutput,
	size_t nTheta, size_t nPhi, size_t nPitch,
	fReal phiOffset, fReal thetaOffset, fReal gridLen,
	fReal radius, fReal timeStep, fReal phiNorm, fReal thetaNorm);*/

void KaminoSolver::advection()
{
	//bindVelocity2Tex(texVelPhi, texVelTheta);
	velPhi->bindTexture(texVelPhi);
	velTheta->bindTexture(texVelTheta);
	
	///kernel call goes here
	// Advect Phi
	velPhi->bindTexture(texBeingAdvected);
	fReal phiNorm = M_2PI;
	fReal thetaNorm = M_PI;
	dim3 gridLayout = dim3(velPhi->getNTheta());
	dim3 blockLayout = dim3(velPhi->getNPhi());
	advectionVPhiKernel<<<gridLayout, blockLayout>>>
	(velPhi->getGPUNextStep(), velPhi->getNTheta(), velPhi->getNPhi(), velPhi->getNextStepPitch(),
	gridLen, radius, timeStep);
	velPhi->unbindTexture(texBeingAdvected);

	// Advect Theta
	velTheta->bindTexture(texBeingAdvected);
	//texBeingAdvected = texVelTheta;
	phiNorm = M_2PI;
	thetaNorm = M_PI - 2 * gridLen;
	gridLayout = dim3(velTheta->getNTheta());
	blockLayout = dim3(velTheta->getNPhi());
	advectionVThetaKernel<<<gridLayout, blockLayout>>>
	(velTheta->getGPUNextStep(), velTheta->getNTheta(), velTheta->getNPhi(), velTheta->getNextStepPitch(),
	gridLen, radius, timeStep);
	velTheta->unbindTexture(texBeingAdvected);

	velPhi->unbindTexture(texVelPhi);
	velTheta->unbindTexture(texVelTheta);

	swapAttrBuffers();
}